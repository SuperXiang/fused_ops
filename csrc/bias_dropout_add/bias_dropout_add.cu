#include "hip/hip_runtime.h"
#include <vector>
#include <ATen/ATen.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <c10/cuda/CUDAMathCompat.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "THC/THC.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>

#define CELL(a, b) (((a) + (b) - 1) / (b))
#if __cplusplus >= 201703L
    #define IF_CONSTEXPR constexpr
#else
    #define IF_CONSTEXPR
#endif

template <typename MaskType, typename acc_t, typename IndexType>
__global__ void generate_dropout_mask_kernel(MaskType* output, IndexType n, uint64_t seed, uint64_t offset, acc_t p) {
    IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, idx, offset, &state);
    MaskType mask = 0;
    #pragma unroll
    for (int i = 0; i < sizeof(MaskType) * 2; ++i) {
        float4 rand = hiprand_uniform4(&state);
        mask |= ((rand.x < p) << (i * 4))
            | ((rand.y < p) << (i * 4 + 1))
            | ((rand.z < p) << (i * 4 + 2))
            | ((rand.w < p) << (i * 4 + 3));
    }
    if (idx < n) {
        output[idx] = mask;
    }
}

template <typename MaskType>
void generate_dropout_mask(MaskType* mask, int bsz, int dim, float p, uint64_t seed, uint64_t offset) {
    int mask_elements_per_batch = CELL(dim, sizeof(MaskType) * 8);
    int num_elements = bsz * mask_elements_per_batch;
    const int block_size = 128;
    int grid = CELL(num_elements, block_size);
    generate_dropout_mask_kernel<MaskType, float, size_t><<<grid, block_size>>>(mask, num_elements, seed, offset, p);
}

template <typename T>
__device__ __forceinline__ T from_uint8(uint8_t input) {
    return (T)input;
}

template <>
__device__ __forceinline__ __hip_bfloat16 from_uint8(uint8_t input) {
    return (__hip_bfloat16)(float)input;
}

template <typename index_t, typename input_t, typename output_t, bool is_training>
__global__ void bias_dropout_add_forward(output_t *dst, const input_t *x, const input_t *bias,
    const input_t *residual, const uint8_t *mask, index_t bsz, int dim, input_t pinv) {
    if IF_CONSTEXPR (is_training) {
        int mask_index = blockIdx.x * CELL(dim, 8);
        uint8_t mask_offset = threadIdx.x % 8;
        for (int j = threadIdx.x; j < dim; j += blockDim.x) {
            if (blockIdx.x < bsz) {
                index_t idx = blockIdx.x * dim + j;
                input_t y = x[idx] + bias[j];
                input_t m = from_uint8<input_t>(((mask[mask_index + j / 8] & (1 << mask_offset)) >> mask_offset));
                dst[idx] = y * m * pinv + residual[idx];
            }
        }
    } else {
        for (int j = threadIdx.x; j < dim; j += blockDim.x) {
            if (blockIdx.x < bsz) {
                index_t idx = blockIdx.x * dim + j;
                dst[idx] = x[idx] + bias[j] + residual[idx];
            }
        }
    }
}

template <typename index_t, typename input_t, typename output_t>
__global__ void bias_dropout_add_backward(output_t *dst, const input_t *grad, const uint8_t *mask, index_t bsz, int dim) {
    int mask_index = blockIdx.x * CELL(dim, 8);
    uint8_t mask_offset = threadIdx.x % 8;
    for (int j = threadIdx.x; j < dim; j += blockDim.x) {
        if (blockIdx.x < bsz) {
            index_t idx = blockIdx.x * dim + j;
            uint8_t m = (mask[mask_index + j / 8] & (1 << mask_offset)) >> mask_offset;
            dst[idx] = grad[idx] * from_uint8<input_t>(m);
        }
    }
}

std::vector<torch::Tensor> bias_dropout_add_forward_cuda(const torch::Tensor &x, const torch::Tensor &bias,
    const torch::Tensor &residual, bool is_training, float dropout_prob, c10::optional<at::Generator> gen_) {
    using MaskType = uint64_t;
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    auto sizes = x.sizes();
    size_t bsz = 1;
    for (size_t i = 0; i + 1 < sizes.size(); ++i) {
        bsz *= sizes[i];
    }
    int dim = sizes[sizes.size() - 1];
    auto dst_options = x.options().requires_grad(false);
    auto mask_options = dst_options.dtype(torch::kInt64);
    torch::Tensor mask = torch::empty(bsz * CELL(dim, sizeof(MaskType) * 8), mask_options);
    torch::Tensor results = torch::empty(sizes, dst_options);
    auto type = x.scalar_type();
    const int ThreadsPerBlock = 256;
    if (is_training) {
        auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(gen_, at::cuda::detail::getDefaultCUDAGenerator());
        std::pair<uint64_t, uint64_t> rng_engine_inputs;
        {
            // See Note [Acquire lock when using random generators]
            std::lock_guard<std::mutex> lock(gen->mutex_);
            rng_engine_inputs = gen->philox_engine_inputs(sizeof(MaskType) * 8);
        }
        uint64_t seed = std::get<0>(rng_engine_inputs);
        uint64_t offset = std::get<1>(rng_engine_inputs);
        generate_dropout_mask<MaskType>((MaskType *)mask.data_ptr(), bsz, dim, 1.0 - dropout_prob, seed, offset);
        if (type == at::ScalarType::BFloat16) {
            bias_dropout_add_forward<size_t, hip_bfloat16, hip_bfloat16, true><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (hip_bfloat16 *)results.data_ptr(),
                (const hip_bfloat16 *)x.data_ptr(),
                (const hip_bfloat16 *)bias.data_ptr(),
                (const hip_bfloat16 *)residual.data_ptr(),
                (const uint8_t *)mask.data_ptr(),
                bsz,
                dim,
                1.0 / (1.0 - dropout_prob));
        } else if (type == at::ScalarType::Half) {
            bias_dropout_add_forward<size_t, half, half, true><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (half *)results.data_ptr(),
                (const half *)x.data_ptr(),
                (const half *)bias.data_ptr(),
                (const half *)residual.data_ptr(),
                (const uint8_t *)mask.data_ptr(),
                bsz,
                dim,
                1.0 / (1.0 - dropout_prob));
        } else if (type == at::ScalarType::Float) {
            bias_dropout_add_forward<size_t, float, float, true><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (float *)results.data_ptr(),
                (const float *)x.data_ptr(),
                (const float *)bias.data_ptr(),
                (const float *)residual.data_ptr(),
                (const uint8_t *)mask.data_ptr(),
                bsz,
                dim,
                1.0 / (1.0 - dropout_prob));
        }
    } else {
        if (type == at::ScalarType::BFloat16) {
            bias_dropout_add_forward<size_t, hip_bfloat16, hip_bfloat16, false><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (hip_bfloat16 *)results.data_ptr(),
                (const hip_bfloat16 *)x.data_ptr(),
                (const hip_bfloat16 *)bias.data_ptr(),
                (const hip_bfloat16 *)residual.data_ptr(),
                nullptr,
                bsz,
                dim,
                0.0);
        } else if (type == at::ScalarType::Half) {
            bias_dropout_add_forward<size_t, half, half, false><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (half *)results.data_ptr(),
                (const half *)x.data_ptr(),
                (const half *)bias.data_ptr(),
                (const half *)residual.data_ptr(),
                nullptr,
                bsz,
                dim,
                0.0);
        } else if (type == at::ScalarType::Float) {
            bias_dropout_add_forward<size_t, float, float, false><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (float *)results.data_ptr(),
                (const float *)x.data_ptr(),
                (const float *)bias.data_ptr(),
                (const float *)residual.data_ptr(),
                nullptr,
                bsz,
                dim,
                0.0);
        }
    }
    return {results, mask};
}

torch::Tensor bias_dropout_add_backward_cuda(const torch::Tensor &grad, const torch::Tensor &mask) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    auto sizes = grad.sizes();
    size_t bsz = 1;
    for (size_t i = 0; i + 1 < sizes.size(); ++i) {
        bsz *= sizes[i];
    }
    int dim = sizes[sizes.size() - 1];
    auto dst_options = grad.options().requires_grad(false);
    torch::Tensor results = torch::empty(sizes, dst_options);
    auto type = grad.scalar_type();
    const int ThreadsPerBlock = 256;
    if (type == at::ScalarType::BFloat16) {
        bias_dropout_add_backward<size_t, hip_bfloat16, hip_bfloat16><<<bsz, ThreadsPerBlock, 0, stream>>>(
            (hip_bfloat16 *)results.data_ptr(),
            (const hip_bfloat16 *)grad.data_ptr(),
            (const uint8_t *)mask.data_ptr(),
            bsz,
            dim);
    } else if (type == at::ScalarType::Half) {
        bias_dropout_add_backward<size_t, half, half><<<bsz, ThreadsPerBlock, 0, stream>>>(
            (half *)results.data_ptr(),
            (const half *)grad.data_ptr(),
            (const uint8_t *)mask.data_ptr(),
            bsz,
            dim);
    } else if (type == at::ScalarType::Float) {
        bias_dropout_add_backward<size_t, float, float><<<bsz, ThreadsPerBlock, 0, stream>>>(
            (float *)results.data_ptr(),
            (const float *)grad.data_ptr(),
            (const uint8_t *)mask.data_ptr(),
            bsz,
            dim);
    }
    return results;
}