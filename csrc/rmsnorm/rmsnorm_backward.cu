#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/cuda/HIPContext.h"
#include <THC/THCDeviceUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include "type_shim.h"

namespace {
// This is the un-specialized struct.  Note that we prevent instantiation of this
// struct by putting an undefined symbol in the function body so it won't compile.
//  template <typename T>
//  struct SharedMemory
//  {
//      // Ensure that we won't compile any un-specialized types
//      __device__ T *getPointer()
//      {
//          extern __device__ void error(void);
//          error();
//          return NULL;
//      }
//  };
// https://github.com/NVIDIA/apex/issues/246
template <typename T>
struct SharedMemory;

template <>
struct SharedMemory <float>
{
    __device__ float *getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template <>
struct SharedMemory <double>
{
    __device__ double *getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};
}

template<typename T, typename U> __device__
void cuLoadWriteStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf,
    const T* input,
    const T* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ invvar
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    U curr_invvar = invvar[i1];
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int i2 = i2_off + k;
      const int load_idx = i1*n2+i2;
      const int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        warp_buf[write_idx] = curr_dout * (curr_input) * curr_invvar;
      } else {
        warp_buf[write_idx] = U(0);
      }
    }
  } else {
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      warp_buf[write_idx] = U(0);
    }
  }
}

template<typename T, typename U> __device__
void cuLoadAddStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf,
    const T* input,
    const T* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ invvar
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    U curr_invvar = invvar[i1];
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int i2 = i2_off + k;
      const int load_idx = i1*n2+i2;
      const int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U curr_input = static_cast<U>(input[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        warp_buf[write_idx] += curr_dout * (curr_input) * curr_invvar;
      }
    }
  }
}

template<typename T, typename U> __global__
void cuComputePartGradGamma(
    const T* __restrict__ dout,
    const T* __restrict__ input,
    const int n1,
    const int n2,
    const U* __restrict__ invvar,
    U epsilon,
    U* part_grad_gamma)
{
    const int numsegs_n1 = (n1+blockDim.y*blockDim.y-1) / (blockDim.y*blockDim.y);
    const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
    const int i1_beg = blockIdx.y * segs_per_block * blockDim.y*blockDim.y;
    const int i1_beg_plus_one = (blockIdx.y+1) * segs_per_block * blockDim.y*blockDim.y;
    const int i1_end = i1_beg_plus_one < n1 ? i1_beg_plus_one : n1;
    const int row_stride = blockDim.x+1;
    const int thr_load_col_off = (threadIdx.x*blockDim.y)&(blockDim.x-1);
    const int thr_load_row_off = (threadIdx.x*blockDim.y)/blockDim.x + threadIdx.y*blockDim.y;
    const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
    SharedMemory<U> shared;
    U* buf = shared.getPointer(); // buf has at least blockDim.x * blockDim.y * blockDim.y + (blockDim.y - 1)*(blockDim.x/blockDim.y) elements
    U* warp_buf = (U*)buf;
    // compute partial sums from strided inputs
    // do this to increase number of loads in flight
    cuLoadWriteStridedInputs(i1_beg,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf,input,dout,i1_end,n2,invvar);
    for (int i1_block = i1_beg+blockDim.y*blockDim.y;  i1_block < i1_end;  i1_block+=blockDim.y*blockDim.y) {
      cuLoadAddStridedInputs(i1_block,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf,input,dout,i1_end,n2,invvar);
    }
    __syncthreads();
    // inter-warp reductions
    // sum within each warp
    U acc1 = U(0);
    for (int k = 0;  k < blockDim.y;  ++k) {
      const int row1 = threadIdx.y + k*blockDim.y;
      const int idx1 = row1*row_stride + threadIdx.x;
      acc1 += warp_buf[idx1];
    }
    warp_buf[threadIdx.y*row_stride+threadIdx.x] = acc1;
    __syncthreads();
    // sum all warps
    for (int offset = blockDim.y/2;  offset > 1;  offset /= 2) {
      if (threadIdx.y < offset) {
        const int row1 = threadIdx.y;
        const int row2 = threadIdx.y + offset;
        const int idx1 = row1*row_stride + threadIdx.x;
        const int idx2 = row2*row_stride + threadIdx.x;
        warp_buf[idx1] += warp_buf[idx2];
      }
      __syncthreads();
    }
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.y == 0 && i2 < n2) {
      const int row1 = threadIdx.y;
      const int row2 = threadIdx.y + 1;
      const int idx1 = row1*row_stride + threadIdx.x;
      const int idx2 = row2*row_stride + threadIdx.x;
      part_grad_gamma[blockIdx.y*n2+i2] = warp_buf[idx1] + warp_buf[idx2];
    }
}

template<typename T, typename U> __global__
void cuComputeGradGamma(
    const U* part_grad_gamma,
    const int part_size,
    const int n1,
    const int n2,
    T* grad_gamma)
{
    // sum partial gradients for gamma and beta
    SharedMemory<U> shared;
    U* buf = shared.getPointer(); 
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (i2 < n2) {
      // each warp does sequential reductions until reduced part_size is num_warps
      int num_warp_reductions = part_size / blockDim.y;
      U sum_gamma = U(0);
      const U* part_grad_gamma_ptr = part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
      for (int warp_offset = 0;  warp_offset < num_warp_reductions;  ++warp_offset) {
        sum_gamma += part_grad_gamma_ptr[warp_offset*n2];
      }
      for (int offset = blockDim.y/2;  offset >= 1;  offset /= 2) {
        // top half write to shared memory
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[write_idx] = sum_gamma;
        }
        __syncthreads();
        // bottom half sums
        if (threadIdx.y < offset) {
          const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
          sum_gamma += buf[read_idx];
        }
        __syncthreads();
      }
      // write out fully summed gradients
      if (threadIdx.y == 0) {
        grad_gamma[i2] = sum_gamma;
      }
    }
}

template<typename T, typename U> 
void HostRMSNormGradient(
    const T* dout,
    const U* invvar,
    at::Tensor* input,
    int n1,
    int n2,
    const T* gamma,
    double epsilon,
    T* grad_gamma
    )
{
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    if (gamma != NULL) {
      // compute grad_gamma(j) and grad_beta(j)
      const int part_size = 16;
      const dim3 threads2(32,4,1);
      const dim3 blocks2((n2+threads2.x-1)/threads2.x,part_size,1);
      const int nshared2_a = 2 * sizeof(U) * threads2.y * threads2.y * (threads2.x + 1);
      const int nshared2_b = threads2.x * threads2.y * sizeof(U);
      const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;
      at::Tensor part_grad_gamma = at::empty({part_size,n2}, input->options().dtype((input->scalar_type()==at::ScalarType::Half || input->scalar_type()==at::ScalarType::BFloat16) ? at::ScalarType::Float : input->scalar_type()));
      cuComputePartGradGamma<<<blocks2, threads2, nshared2, stream>>>(
              dout,
              input->data_ptr<T>(),
              n1,n2,
              invvar,
              U(epsilon),
              part_grad_gamma.data_ptr<U>());

      const dim3 threads3(32,8,1);
      const dim3 blocks3((n2+threads2.x-1)/threads2.x,1,1);
      const int nshared3 = threads3.x * threads3.y * sizeof(U);
      cuComputeGradGamma<<<blocks3, threads3, nshared3, stream>>>(
              part_grad_gamma.data_ptr<U>(),
              part_size,
              n1,n2,
              grad_gamma);
    }
}

void cuda_rms_norm_gradient(
    at::Tensor* dout,
    at::Tensor* invvar,
    at::Tensor* input,
    int n1,
    int n2,
    at::IntArrayRef normalized_shape,
    at::Tensor* gamma,
    double epsilon,
    at::Tensor* grad_gamma)
{
    using namespace at;
    DISPATCH_DOUBLE_FLOAT_AND_HALF_AND_BF16(input->scalar_type(), 0, "cuComputeGradInput",
        using accscalar_t = at::acc_type<scalar_t_0, true>;
        HostRMSNormGradient(
        dout->data_ptr<scalar_t_0>(),
        invvar->data_ptr<accscalar_t>(),
        input,
        n1,n2,
        gamma->data_ptr<scalar_t_0>(),
        epsilon,
        grad_gamma->data_ptr<scalar_t_0>());
      )
}